// residual.cu — Residual Connection Kernel - LT-014
//
// Implements residual connections for Llama transformer blocks.
// output = input + residual (element-wise addition)
//
// Spec: M0-W-1214

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cstdint>
#include <stdio.h>

/**
 * Residual connection kernel - element-wise addition
 * 
 * Formula:
 *   output[i] = input[i] + residual[i]
 */
__global__ void residual_kernel(
    half* output,
    const half* input,
    const half* residual,
    int total_elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < total_elements) {
        output[idx] = __hadd(input[idx], residual[idx]);
    }
}

/**
 * Vectorized residual kernel using half2 for better throughput
 */
__global__ void residual_kernel_vectorized(
    half2* output,
    const half2* input,
    const half2* residual,
    int total_elements_half2
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < total_elements_half2) {
        output[idx] = __hadd2(input[idx], residual[idx]);
    }
}

extern "C" {

/**
 * Apply residual connection
 * 
 * @param output Output tensor [batch, seq_len, hidden_dim]
 * @param input Input tensor [batch, seq_len, hidden_dim]
 * @param residual Residual tensor [batch, seq_len, hidden_dim]
 * @param batch_size Batch size
 * @param seq_len Sequence length
 * @param hidden_dim Hidden dimension
 * @param in_place If true, output += residual (input ignored)
 * @return 0 on success, error code on failure
 */
int cuda_residual_forward(
    half* output,
    const half* input,
    const half* residual,
    int batch_size,
    int seq_len,
    int hidden_dim,
    bool in_place
) {
    // Validate dimensions
    if (batch_size <= 0 || seq_len <= 0 || hidden_dim <= 0) {
        fprintf(stderr, "Residual: Invalid dimensions\n");
        return -1;
    }
    
    int total_elements = batch_size * seq_len * hidden_dim;
    
    // Use vectorized kernel if hidden_dim is even
    if (hidden_dim % 2 == 0) {
        int total_half2 = total_elements / 2;
        int threads = 256;
        int blocks = (total_half2 + threads - 1) / threads;
        
        if (in_place) {
            residual_kernel_vectorized<<<blocks, threads>>>(
                (half2*)output,
                (const half2*)output,
                (const half2*)residual,
                total_half2
            );
        } else {
            residual_kernel_vectorized<<<blocks, threads>>>(
                (half2*)output,
                (const half2*)input,
                (const half2*)residual,
                total_half2
            );
        }
    } else {
        // Fall back to non-vectorized kernel
        int threads = 256;
        int blocks = (total_elements + threads - 1) / threads;
        
        if (in_place) {
            residual_kernel<<<blocks, threads>>>(
                output,
                output,
                residual,
                total_elements
            );
        } else {
            residual_kernel<<<blocks, threads>>>(
                output,
                input,
                residual,
                total_elements
            );
        }
    }
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Residual kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    return 0;
}

/**
 * Wrapper for transformer compatibility
 * Maps cuda_residual_add to cuda_residual_forward
 */
void cuda_residual_add(
    const void* input,
    const void* residual,
    void* output,
    uint32_t batch_size,
    uint32_t hidden_dim,
    hipStream_t stream
) {
    // Calculate total elements
    int total_elements = batch_size * hidden_dim;
    
    // Cast to half pointers
    const half* input_half = reinterpret_cast<const half*>(input);
    const half* residual_half = reinterpret_cast<const half*>(residual);
    half* output_half = reinterpret_cast<half*>(output);
    
    // Use vectorized kernel if possible
    if (hidden_dim % 2 == 0) {
        int total_half2 = total_elements / 2;
        int threads = 256;
        int blocks = (total_half2 + threads - 1) / threads;
        
        residual_kernel_vectorized<<<blocks, threads, 0, stream>>>(
            (half2*)output_half,
            (const half2*)input_half,
            (const half2*)residual_half,
            total_half2
        );
    } else {
        int threads = 256;
        int blocks = (total_elements + threads - 1) / threads;
        
        residual_kernel<<<blocks, threads, 0, stream>>>(
            output_half,
            input_half,
            residual_half,
            total_elements
        );
    }
}

} // extern "C"
