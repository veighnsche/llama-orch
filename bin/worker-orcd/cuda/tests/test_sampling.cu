#include "hip/hip_runtime.h"
/**
 * Sampling Kernels Unit Tests
 * 
 * Tests temperature scaling and greedy sampling kernel correctness.
 * 
 * Spec: M0-W-1032, M0-W-1421, KERNEL-SAMPLE-003
 * Story: FT-017, FT-018
 */

#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "../kernels/sampling.cuh"
#include <vector>
#include <cmath>

using namespace worker::kernels;

// ============================================================================
// Test Fixture
// ============================================================================

class TemperatureScaleTest : public ::testing::Test {
protected:
    void SetUp() override {
        // Initialize CUDA device
        int device_count;
        hipGetDeviceCount(&device_count);
        if (device_count == 0) {
            GTEST_SKIP() << "No CUDA devices available";
        }
        hipSetDevice(0);
        
        // Allocate device memory
        vocab_size = 1000;
        hipMalloc(&d_logits_fp32, vocab_size * sizeof(float));
        hipMalloc(&d_logits_fp16, vocab_size * sizeof(half));
    }
    
    void TearDown() override {
        hipFree(d_logits_fp32);
        hipFree(d_logits_fp16);
    }
    
    int vocab_size;
    float* d_logits_fp32 = nullptr;
    half* d_logits_fp16 = nullptr;
};

// ============================================================================
// Basic Functionality Tests (FP32)
// ============================================================================

/**
 * Test: Temperature = 1.0 (no change)
 * 
 * Spec: M0-W-1032 (Temperature Scaling)
 * Critical: Identity case must work
 */
TEST_F(TemperatureScaleTest, TemperatureOneNoChange) {
    // Temperature = 1.0 should not change logits
    std::vector<float> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = static_cast<float>(i);
    }
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, 1.0f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify unchanged
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_FLOAT_EQ(h_result[i], h_logits[i]) << "Index " << i;
    }
}

/**
 * Test: Temperature = 0.5 (doubles logits)
 * 
 * Spec: M0-W-1032 (Temperature Scaling)
 * Critical: Scaling math must be correct
 */
TEST_F(TemperatureScaleTest, TemperatureHalfDoublesLogits) {
    // Temperature = 0.5 should double logits (divide by 0.5 = multiply by 2)
    std::vector<float> h_logits(vocab_size, 1.0f);
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, 0.5f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify doubled
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_NEAR(h_result[i], 2.0f, 0.001f) << "Index " << i;
    }
}

/**
 * Test: Temperature = 2.0 (halves logits)
 * 
 * Spec: M0-W-1032 (Temperature Scaling)
 * Critical: Scaling math must be correct
 */
TEST_F(TemperatureScaleTest, TemperatureTwoHalvesLogits) {
    // Temperature = 2.0 should halve logits
    std::vector<float> h_logits(vocab_size, 4.0f);
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, 2.0f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify halved
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_NEAR(h_result[i], 2.0f, 0.001f) << "Index " << i;
    }
}

/**
 * Test: Temperature = 0.0 (greedy mode, no change)
 * 
 * Spec: M0-W-1032 (Temperature Scaling)
 * Critical: Special case for testing reproducibility
 */
TEST_F(TemperatureScaleTest, TemperatureZeroNoChange) {
    // Temperature = 0.0 should not change logits (greedy mode)
    std::vector<float> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = static_cast<float>(i);
    }
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, 0.0f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify unchanged (greedy mode)
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_FLOAT_EQ(h_result[i], h_logits[i]) << "Index " << i;
    }
}

/**
 * Test: Negative logits
 * 
 * Critical: Handles negative values correctly
 */
TEST_F(TemperatureScaleTest, NegativeLogits) {
    // Test with negative logits
    std::vector<float> h_logits(vocab_size, -2.0f);
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, 0.5f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // -2.0 / 0.5 = -4.0
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_NEAR(h_result[i], -4.0f, 0.001f) << "Index " << i;
    }
}

/**
 * Test: Large vocabulary (Qwen 151936)
 * 
 * Critical: Real vocabulary size
 */
TEST_F(TemperatureScaleTest, LargeVocabulary) {
    // Test with realistic vocabulary size (Qwen2.5-0.5B)
    int large_vocab = 151936;
    float* d_large_logits;
    hipMalloc(&d_large_logits, large_vocab * sizeof(float));
    
    std::vector<float> h_logits(large_vocab, 1.0f);
    hipMemcpy(d_large_logits, h_logits.data(), large_vocab * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_large_logits, large_vocab, 0.7f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(large_vocab);
    hipMemcpy(h_result.data(), d_large_logits, large_vocab * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // 1.0 / 0.7 = 1.428571...
    float expected = 1.0f / 0.7f;
    for (int i = 0; i < large_vocab; ++i) {
        EXPECT_NEAR(h_result[i], expected, 0.01f) << "Index " << i;
    }
    
    hipFree(d_large_logits);
}

// ============================================================================
// FP16 Tests
// ============================================================================

/**
 * Test: FP16 temperature scaling
 * 
 * Critical: FP16 variant must work correctly
 */
TEST_F(TemperatureScaleTest, FP16Scaling) {
    std::vector<half> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = __float2half(2.0f);
    }
    
    hipMemcpy(d_logits_fp16, h_logits.data(), vocab_size * sizeof(half), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp16(d_logits_fp16, vocab_size, 0.5f);
    hipDeviceSynchronize();
    
    std::vector<half> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp16, vocab_size * sizeof(half), 
               hipMemcpyDeviceToHost);
    
    // 2.0 / 0.5 = 4.0
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_NEAR(__half2float(h_result[i]), 4.0f, 0.1f) << "Index " << i;
    }
}

/**
 * Test: FP16 temperature = 0.0 (greedy mode)
 * 
 * Critical: FP16 greedy mode must work
 */
TEST_F(TemperatureScaleTest, FP16TemperatureZero) {
    std::vector<half> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = __float2half(static_cast<float>(i));
    }
    
    hipMemcpy(d_logits_fp16, h_logits.data(), vocab_size * sizeof(half), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp16(d_logits_fp16, vocab_size, 0.0f);
    hipDeviceSynchronize();
    
    std::vector<half> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp16, vocab_size * sizeof(half), 
               hipMemcpyDeviceToHost);
    
    // Verify unchanged
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_NEAR(__half2float(h_result[i]), __half2float(h_logits[i]), 0.01f) << "Index " << i;
    }
}

// ============================================================================
// Edge Case Tests
// ============================================================================

/**
 * Test: Invalid temperature (negative)
 * 
 * Critical: Defensive programming
 */
TEST_F(TemperatureScaleTest, InvalidTemperatureNegative) {
    std::vector<float> h_logits(vocab_size, 1.0f);
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    // Negative temperature should be ignored (no scaling)
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, -0.5f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify unchanged (invalid temperature ignored)
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_FLOAT_EQ(h_result[i], 1.0f) << "Index " << i;
    }
}

/**
 * Test: Invalid temperature (too large)
 * 
 * Critical: Defensive programming
 */
TEST_F(TemperatureScaleTest, InvalidTemperatureTooLarge) {
    std::vector<float> h_logits(vocab_size, 1.0f);
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    // Temperature > 2.0 should be ignored (no scaling)
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, 3.0f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify unchanged (invalid temperature ignored)
    for (int i = 0; i < vocab_size; ++i) {
        EXPECT_FLOAT_EQ(h_result[i], 1.0f) << "Index " << i;
    }
}

/**
 * Test: Mixed positive and negative logits
 * 
 * Critical: Realistic logit distribution
 */
TEST_F(TemperatureScaleTest, MixedLogits) {
    std::vector<float> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = static_cast<float>(i - 500);  // Range: -500 to 499
    }
    
    hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    float temperature = 0.8f;
    launch_temperature_scale_fp32(d_logits_fp32, vocab_size, temperature);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(vocab_size);
    hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify scaling
    for (int i = 0; i < vocab_size; ++i) {
        float expected = h_logits[i] / temperature;
        EXPECT_NEAR(h_result[i], expected, 0.01f) << "Index " << i;
    }
}

// ============================================================================
// Real-World Scenarios
// ============================================================================

/**
 * Test: Common temperature values
 * 
 * Critical: Production temperature ranges
 */
TEST_F(TemperatureScaleTest, CommonTemperatureValues) {
    std::vector<float> temperatures = {0.0f, 0.3f, 0.5f, 0.7f, 0.9f, 1.0f, 1.2f, 1.5f, 2.0f};
    
    for (float temp : temperatures) {
        std::vector<float> h_logits(vocab_size, 1.0f);
        
        hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
                   hipMemcpyHostToDevice);
        
        launch_temperature_scale_fp32(d_logits_fp32, vocab_size, temp);
        hipDeviceSynchronize();
        
        std::vector<float> h_result(vocab_size);
        hipMemcpy(h_result.data(), d_logits_fp32, vocab_size * sizeof(float), 
                   hipMemcpyDeviceToHost);
        
        // Verify scaling
        float expected = (temp == 0.0f) ? 1.0f : (1.0f / temp);
        EXPECT_NEAR(h_result[0], expected, 0.01f) << "Temperature " << temp;
    }
}

/**
 * Test: GPT-OSS-20B vocabulary (50257)
 * 
 * Critical: Real model vocabulary
 */
TEST_F(TemperatureScaleTest, GPTVocabulary) {
    int gpt_vocab = 50257;
    float* d_gpt_logits;
    hipMalloc(&d_gpt_logits, gpt_vocab * sizeof(float));
    
    std::vector<float> h_logits(gpt_vocab);
    for (int i = 0; i < gpt_vocab; ++i) {
        h_logits[i] = static_cast<float>(i % 100) * 0.1f;
    }
    
    hipMemcpy(d_gpt_logits, h_logits.data(), gpt_vocab * sizeof(float), 
               hipMemcpyHostToDevice);
    
    launch_temperature_scale_fp32(d_gpt_logits, gpt_vocab, 0.9f);
    hipDeviceSynchronize();
    
    std::vector<float> h_result(gpt_vocab);
    hipMemcpy(h_result.data(), d_gpt_logits, gpt_vocab * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Verify scaling
    for (int i = 0; i < gpt_vocab; ++i) {
        float expected = h_logits[i] / 0.9f;
        EXPECT_NEAR(h_result[i], expected, 0.01f) << "Index " << i;
    }
    
    hipFree(d_gpt_logits);
}

// ============================================================================
// Determinism Tests
// ============================================================================

/**
 * Test: Temperature scaling is deterministic
 * 
 * Property: Same inputs → same outputs (every time)
 */
TEST_F(TemperatureScaleTest, DeterministicScaling) {
    std::vector<float> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = static_cast<float>(i % 100) * 0.1f;
    }
    
    float temperature = 0.7f;
    
    // Run kernel 5 times
    std::vector<std::vector<float>> results(5);
    for (int run = 0; run < 5; ++run) {
        // Reset logits
        hipMemcpy(d_logits_fp32, h_logits.data(), vocab_size * sizeof(float), 
                   hipMemcpyHostToDevice);
        
        launch_temperature_scale_fp32(d_logits_fp32, vocab_size, temperature);
        hipDeviceSynchronize();
        
        results[run].resize(vocab_size);
        hipMemcpy(results[run].data(), d_logits_fp32, vocab_size * sizeof(float), 
                   hipMemcpyDeviceToHost);
    }
    
    // Verify all runs produce identical results
    for (int run = 1; run < 5; ++run) {
        for (int i = 0; i < vocab_size; ++i) {
            EXPECT_FLOAT_EQ(results[0][i], results[run][i]) 
                << "Run " << run << ", index " << i << " differs";
        }
    }
}

// ============================================================================
// Greedy Sampling Tests
// ============================================================================

class GreedySamplingTest : public ::testing::Test {
protected:
    void SetUp() override {
        // Initialize CUDA device
        int device_count;
        hipGetDeviceCount(&device_count);
        if (device_count == 0) {
            GTEST_SKIP() << "No CUDA devices available";
        }
        hipSetDevice(0);
    }
};

/**
 * Test: Simple argmax (token in middle)
 * 
 * Spec: M0-W-1421 (Token Sampling)
 * Critical: Core argmax must work
 */
TEST_F(GreedySamplingTest, SimpleArgmax) {
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size, 0.0f);
    
    // Set token 500 to highest value
    h_logits[500] = 10.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 500);
    
    hipFree(d_logits);
}

/**
 * Test: First token is max
 * 
 * Critical: Edge case (first element)
 */
TEST_F(GreedySamplingTest, FirstToken) {
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size, 0.0f);
    
    // Set token 0 to highest value
    h_logits[0] = 10.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 0);
    
    hipFree(d_logits);
}

/**
 * Test: Last token is max
 * 
 * Critical: Edge case (last element)
 */
TEST_F(GreedySamplingTest, LastToken) {
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size, 0.0f);
    
    // Set last token to highest value
    h_logits[vocab_size - 1] = 10.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, vocab_size - 1);
    
    hipFree(d_logits);
}

/**
 * Test: Negative logits
 * 
 * Critical: Handles negative values correctly
 */
TEST_F(GreedySamplingTest, NegativeLogits) {
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size, -10.0f);
    
    // Set token 250 to least negative (highest)
    h_logits[250] = -1.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 250);
    
    hipFree(d_logits);
}

/**
 * Test: Large vocabulary (Qwen 151936)
 * 
 * Spec: M0-W-1421 (Token Sampling)
 * Critical: Real vocabulary size
 */
TEST_F(GreedySamplingTest, LargeVocabulary) {
    // Test with Qwen vocabulary size
    int vocab_size = 151936;
    std::vector<float> h_logits(vocab_size, 0.0f);
    
    // Set token 100000 to highest value
    h_logits[100000] = 10.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 100000);
    
    hipFree(d_logits);
}

/**
 * Test: Determinism (multiple runs → same result)
 * 
 * Spec: M0-W-1032 (Temperature Scaling)
 * Critical: Greedy sampling must be deterministic
 */
TEST_F(GreedySamplingTest, Determinism) {
    // Test that greedy sampling is deterministic
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size);
    
    // Random logits
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = static_cast<float>(i % 100);
    }
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    // Run multiple times
    int token_id1 = launch_greedy_sample(d_logits, vocab_size);
    int token_id2 = launch_greedy_sample(d_logits, vocab_size);
    int token_id3 = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id1, token_id2);
    EXPECT_EQ(token_id2, token_id3);
    
    hipFree(d_logits);
}

/**
 * Test: Multiple peaks (tie-breaking)
 * 
 * Critical: Consistent tie-breaking behavior
 */
TEST_F(GreedySamplingTest, MultiplePeaks) {
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size, 0.0f);
    
    // Set multiple tokens to same max value
    h_logits[100] = 10.0f;
    h_logits[500] = 10.0f;
    h_logits[800] = 10.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    // Should consistently pick one of them
    int token_id1 = launch_greedy_sample(d_logits, vocab_size);
    int token_id2 = launch_greedy_sample(d_logits, vocab_size);
    
    // Verify it's one of the max values
    EXPECT_TRUE(token_id1 == 100 || token_id1 == 500 || token_id1 == 800);
    // Verify determinism (same result every time)
    EXPECT_EQ(token_id1, token_id2);
    
    hipFree(d_logits);
}

/**
 * Test: Small vocabulary
 * 
 * Critical: Works with small vocab sizes
 */
TEST_F(GreedySamplingTest, SmallVocabulary) {
    int vocab_size = 10;
    std::vector<float> h_logits = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 
                                     6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 9);  // Last element is max
    
    hipFree(d_logits);
}

/**
 * Test: GPT-OSS-20B vocabulary (50257)
 * 
 * Critical: Real model vocabulary
 */
TEST_F(GreedySamplingTest, GPTVocabulary) {
    int vocab_size = 50257;
    std::vector<float> h_logits(vocab_size, 0.0f);
    
    // Set token 25000 to highest value
    h_logits[25000] = 5.0f;
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 25000);
    
    hipFree(d_logits);
}

/**
 * Test: Mixed positive and negative logits
 * 
 * Critical: Realistic logit distribution
 */
TEST_F(GreedySamplingTest, MixedLogits) {
    int vocab_size = 1000;
    std::vector<float> h_logits(vocab_size);
    for (int i = 0; i < vocab_size; ++i) {
        h_logits[i] = static_cast<float>(i - 500);  // Range: -500 to 499
    }
    
    float* d_logits;
    hipMalloc(&d_logits, vocab_size * sizeof(float));
    hipMemcpy(d_logits, h_logits.data(), vocab_size * sizeof(float), 
               hipMemcpyHostToDevice);
    
    int token_id = launch_greedy_sample(d_logits, vocab_size);
    
    EXPECT_EQ(token_id, 999);  // Last element is max (499)
    
    hipFree(d_logits);
}

/**
 * Test: Error handling - invalid vocab_size
 * 
 * Critical: Defensive programming
 */
TEST_F(GreedySamplingTest, InvalidVocabSize) {
    float* d_logits;
    hipMalloc(&d_logits, 1000 * sizeof(float));
    
    // Test with invalid vocab_size
    int token_id = launch_greedy_sample(d_logits, 0);
    EXPECT_EQ(token_id, -1);
    
    token_id = launch_greedy_sample(d_logits, -100);
    EXPECT_EQ(token_id, -1);
    
    hipFree(d_logits);
}

/**
 * Test: Error handling - null pointer
 * 
 * Critical: Defensive programming
 */
TEST_F(GreedySamplingTest, NullPointer) {
    int token_id = launch_greedy_sample(nullptr, 1000);
    EXPECT_EQ(token_id, -1);
}

// ---
// Built by Foundation-Alpha 🏗️
